#include "hip/hip_runtime.h"
/*!
 * Copyright (c) 2016 by Contributors
 * \file permutohedral.cu
 * \brief
 * \author Junyuan Xie
*/

#include "./permutohedral-inl.h"
#include "cu_hash_table.h"

namespace mxnet {
namespace op {

namespace permutohedral {

template<int key_size>
__global__ void init(CuHashTable<key_size> table,
                     const int n_elements,
                     const float *pos,
                     const float *scale,
                     Pair *matrix) {
  float elevated[key_size+1];
  int greedy[key_size+1];
  int rank[key_size+1];
  float barycentric[key_size+2];
  short key[key_size];

  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_elements) return;

  float sm = 0;
  for (int i = key_size; i > 0; i--) {
    float cf = pos[(i-1)*n_elements + idx]*scale[i-1];
    elevated[i] = sm - i*cf;
    sm += cf;
  }
  elevated[0] = sm;  
    
  // find the closest zero-colored lattice point

  // greedily search for the closest zero-colored lattice point
  short sum = 0;
  for (int i = 0; i <= key_size; i++) {
    float v = elevated[i]*(1.0f/(key_size+1));
    float up = ceilf(v) * (key_size+1);
    float down = floorf(v) * (key_size+1);
    if (up - elevated[i] < elevated[i] - down) {
      greedy[i] = static_cast<short>(up);
    } else {
      greedy[i] = static_cast<short>(down);
    }
    sum += greedy[i];
  }
  sum /= key_size+1;
  
  // sort differential to find the permutation between this simplex and the canonical one
  for (int i = 0; i <= key_size; i++) {
    rank[i] = 0;
    for (int j = 0; j <= key_size; j++) {
      if (elevated[i] - greedy[i] < elevated[j] - greedy[j] ||
          (elevated[i] - greedy[i] == elevated[j] - greedy[j]
           && i > j)) {
        rank[i]++;
      }
    }
  }
  
  if (sum > 0) { // sum too large, need to bring down the ones with the smallest differential
    for (int i = 0; i <= key_size; i++) {
      if (rank[i] >= key_size + 1 - sum) {
        greedy[i] -= key_size+1;
        rank[i] += sum - (key_size+1);
      } else {
        rank[i] += sum;
      }
    }
  } else if (sum < 0) { // sum too small, need to bring up the ones with largest differential
    for (int i = 0; i <= key_size; i++) {
      if (rank[i] < -sum) {
        greedy[i] += key_size+1;
        rank[i] += (key_size+1) + sum;
      } else {
        rank[i] += sum;
      }
    }
  }

  // turn delta into barycentric coords
  for (int i = 0; i <= key_size+1; i++) {
      barycentric[i] = 0;
  }
  
  for (int i = 0; i <= key_size; i++) {
    float delta = (elevated[i] - greedy[i]) * (1.0f/(key_size+1));
    barycentric[key_size-rank[i]] += delta;
    barycentric[key_size+1-rank[i]] -= delta;
  }
  barycentric[0] += 1.0f + barycentric[key_size+1];

  for (int color = 0; color <= key_size; color++) {
    // Compute the location of the lattice point explicitly (all but
    // the last coordinate - it's redundant because they sum to zero)
    for (int i = 0; i < key_size; i++) {
      key[i] = greedy[i] + color;
      if (rank[i] > key_size-color) key[i] -= (key_size+1);
    }

    Pair r;
    r.index = table.insert(key, idx*(key_size+1)+color);
    r.weight = barycentric[color];
    matrix[idx*(key_size+1) + color] = r;
  }
}

template<int key_size, bool norm>
__global__ void splat(CuHashTable<key_size> table,
                      const int32_t n_elements,
                      const int32_t val_size,
                      float *data,
                      float *val,
                      Pair *matrix) {
  const int idx = threadIdx.y + blockIdx.y * blockDim.y;
  if (idx >= n_elements) return;
  const int color = threadIdx.x;

  Pair r = matrix[idx*(key_size+1)+color];
  float *dst = val + r.index*val_size;
  if (!norm) {
    for (int j = 0; j < val_size; j++) {
      atomicAdd(dst+j, data[j*n_elements + idx]*r.weight);
    }
  } else {
    for (int j = 0; j < val_size-1; j++) {
      atomicAdd(dst+j, data[j*n_elements + idx]*r.weight);
    }
    atomicAdd(dst+val_size-1, 1.f*r.weight);
  }
}


template<int key_size>
__global__ static void blur(CuHashTable<key_size> table,
                            const int32_t val_size,
                            const int32_t color,
                            float *val,
                            float *new_val, 
                            Pair *matrix) {
  short key[key_size+1];
  short np[key_size+1];
  short nm[key_size+1];
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx >= table.n_keys_) return;

  // Check if I'm valid
  if (matrix[idx].index != idx) return;

  // find my key and the keys of my neighbours

  for (int i = 0; i < key_size; i++) {
    key[i] = table.keys_[idx*key_size+i];
    np[i] = key[i]+1;    
    nm[i] = key[i]-1;
  }

  np[color] -= key_size+1;
  nm[color] += key_size+1;

  int offNp = table.find(np);
  int offNm = table.find(nm);

  float *valMe = val + val_size*idx;
  float *valNp = val + val_size*offNp;
  float *valNm = val + val_size*offNm; 
  float *valOut = new_val + val_size*idx;

  for (int i = 0; i < val_size; i++) {
    float o = valMe[i];
    if (offNp >= 0) o += 0.5f*valNp[i];
    if (offNm >= 0) o += 0.5f*valNm[i];
    valOut[i] = o;
  }
}

template<int key_size, bool norm>
__global__ void slice(CuHashTable<key_size> table,
                      const int32_t n_elements,
                      const int32_t val_size,
                      float *val,
                      float *out,
                      Pair *matrix) {
  const float alpha = 1.0f / (1+powf(2, -key_size));
  int32_t index[key_size+1];
  float weight[key_size+1];

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;    
  if (idx >= n_elements) return;

  for (int i = 0; i <= key_size; ++i) {
    Pair r = matrix[idx*(key_size+1) + i];
    index[i] = r.index;
    weight[i] = r.weight;
  }

  if (!norm) {
    for (int j = 0; j < val_size; ++j) {
      float v = 0.0f;
      for (int i = 0; i <= key_size; ++i) {
        v += weight[i]*val[index[i]*val_size + j];
      }
      out[j*n_elements + idx] = v * alpha;
    }
  } else {
    float n = 0.0f;
    for (int i = 0; i <= key_size; ++i) {
      n += weight[i]*val[index[i]*val_size + val_size - 1];
    }
    n = 1.0f/n;
    for (int j = 0; j < val_size-1; ++j) {
      float v = 0.0f;
      for (int i = 0; i <= key_size; ++i) {
        v += weight[i]*val[index[i]*val_size + j];
      }
      out[j*n_elements + idx] = v * n;
    }
  }
}

}


template<int key_size>
void CuPermutohedralOp<key_size>::GetTempSpace(const OpContext &ctx) {
  using namespace mshadow;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();

  Tensor<gpu, 1, uint8_t> tmp =
    ctx.requested[kTemp].get_space_typed<gpu, 1, uint8_t>(
      Shape1(n_keys_*2*sizeof(int32_t) +
             n_keys_*key_size*sizeof(int16_t) +
             n_keys_*val_size_*sizeof(float) +
             n_keys_*val_size_*sizeof(float) +
             n_keys_*sizeof(Pair)), s);
  uint8_t *ptr = tmp.dptr_;

  int32_t *entries = (int32_t*)ptr;
  entries_ = Tensor<gpu, 1, int32_t>(entries, Shape1(n_keys_*2), s);
  ptr += n_keys_*2*sizeof(int32_t);

  int16_t *keys = (int16_t*)ptr;
  keys_ = Tensor<gpu, 2, int16_t>(keys, Shape2(key_size, n_keys_), s);
  ptr += n_keys_*key_size*sizeof(int16_t);

  float *vals = (float*)ptr;
  vals_ = Tensor<gpu, 2, float>(vals, Shape2(val_size_, n_keys_), s);
  ptr += n_keys_*val_size_*sizeof(float);

  float *new_vals = (float*)ptr;
  new_vals_ = Tensor<gpu, 2, float>(new_vals, Shape2(val_size_, n_keys_), s);
  ptr += n_keys_*val_size_*sizeof(float);

  Pair *matrix = (Pair*)ptr;
  matrix_ = Tensor<gpu, 1, Pair>(matrix, Shape1(n_keys_), s);
  ptr += n_keys_*sizeof(Pair);

  CHECK_EQ(ptr, tmp.dptr_ + tmp.shape_.Size());
}

template<int key_size>
void CuPermutohedralOp<key_size>::Forward(const OpContext &ctx,
                                   const std::vector<TBlob> &in_data,
                                   const std::vector<OpReqType> &req,
                                   const std::vector<TBlob> &out_data,
                                   const std::vector<TBlob> &aux_args)  {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  Tensor<gpu, 1, float> scale = aux_args[kScale].get<gpu, 1, float>(s);

  if (!init_) {
    TShape data_shape = in_data[kData].shape_;
    batch_size_ = data_shape[0];
    data_size_ = data_shape[1];
    if (param_.normalize) {
      val_size_ = data_size_ + 1;
    } else {
      val_size_ = data_size_;
    }
    n_elements_ = data_shape.Size()/batch_size_/data_size_;
    n_keys_ = n_elements_*(key_size+1);
    CHECK_EQ(in_data[kPos].size(1), key_size);
    
    lblock_ = cuda::kBaseThreadNum;
    nblock_ = (n_elements_-1)/lblock_+1;

    float cpu_scale[key_size];
    for (int i = 0; i < key_size; i++) {
      cpu_scale[i] = (key_size+1)*sqrtf((2.0/3.0)/((i+1)*(i+2)));
    }
    CHECK_EQ(hipMemcpyAsync((void*)scale.dptr_, (void*)cpu_scale, key_size*sizeof(float), hipMemcpyHostToDevice, stream), hipSuccess);

    init_ = true;
  }
  

  Shape<3> shape = Shape3(batch_size_, data_size_, n_elements_); 
  Tensor<gpu, 3, float> in = in_data[kData].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> out = out_data[kOut].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = key_size;
  Tensor<gpu, 3, float> pos = in_data[kPos].get_with_shape<gpu, 3, float>(shape, s);

  GetTempSpace(ctx);

  CuHashTable<key_size> table(n_keys_, entries_.dptr_, keys_.dptr_);


  for (int i = 0; i < batch_size_; ++i) {
    entries_ = -1;
    vals_ = 0;

    init<key_size><<<dim3(nblock_, 1, 1), dim3(lblock_,1,1), 0, stream>>>(
      table, n_elements_, pos.dptr_ + i*key_size*n_elements_, scale.dptr_, matrix_.dptr_);
    CHECK_EQ(hipGetLastError(), hipSuccess);
    if (param_.normalize) {
      splat<key_size, true><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
        table, n_elements_, val_size_, in.dptr_+i*data_size_*n_elements_, vals_.dptr_, matrix_.dptr_);
    } else {
      splat<key_size, false><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
        table, n_elements_, val_size_, in.dptr_+i*data_size_*n_elements_, vals_.dptr_, matrix_.dptr_);
    }
    CHECK_EQ(hipGetLastError(), hipSuccess);

    float *pval = vals_.dptr_;
    float *pnew_val = new_vals_.dptr_;
    for (int j = 0; j <= key_size; ++j) {
      blur<key_size><<<dim3((n_keys_-1)/lblock_+1, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, val_size_, j, pval, pnew_val, matrix_.dptr_);
      CHECK_EQ(hipGetLastError(), hipSuccess);
      std::swap(pval, pnew_val);
    }

    if (param_.normalize) {
      slice<key_size, true><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size_, pval, out.dptr_ + i*data_size_*n_elements_, matrix_.dptr_);
    } else {
      slice<key_size, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size_, pval, out.dptr_ + i*data_size_*n_elements_, matrix_.dptr_);
    }
    CHECK_EQ(hipGetLastError(), hipSuccess);
  }
}

template<int key_size>
void CuPermutohedralOp<key_size>::Backward(const OpContext &ctx,
                                           const std::vector<TBlob> &out_grad,
                                           const std::vector<TBlob> &in_data,
                                           const std::vector<TBlob> &out_data,
                                           const std::vector<OpReqType> &req,
                                           const std::vector<TBlob> &in_grad,
                                           const std::vector<TBlob> &aux_args) {
  using namespace mshadow;
  using namespace mshadow::expr;
  using namespace permutohedral;
  Stream<gpu> *s = ctx.get_stream<gpu>();
  hipStream_t stream = Stream<gpu>::GetStream(s);

  Tensor<gpu, 1, float> scale = aux_args[kScale].get<gpu, 1, float>(s);

  Shape<3> shape = Shape3(batch_size_, data_size_, n_elements_); 
  Tensor<gpu, 3, float> ograd = out_grad[kOut].get_with_shape<gpu, 3, float>(shape, s);
  Tensor<gpu, 3, float> data_grad = in_grad[kData].get_with_shape<gpu, 3, float>(shape, s);
  shape[1] = key_size;
  Tensor<gpu, 3, float> pos = in_data[kPos].get_with_shape<gpu, 3, float>(shape, s);

  GetTempSpace(ctx);

  CuHashTable<key_size> table(n_keys_, entries_.dptr_, keys_.dptr_);

  for (int i = 0; i < batch_size_; ++i) {
    entries_ = -1;
    vals_ = 0;
    
    init<key_size><<<dim3(nblock_, 1, 1), dim3(lblock_,1,1), 0, stream>>>(
      table, n_elements_, pos.dptr_ + i*key_size*n_elements_, scale.dptr_, matrix_.dptr_);
    CHECK_EQ(hipGetLastError(), hipSuccess);
    if (param_.normalize) {
      splat<key_size, true><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
        table, n_elements_, val_size_, ograd.dptr_ + i*data_size_*n_elements_, vals_.dptr_, matrix_.dptr_);
    } else {
      splat<key_size, false><<<dim3(1, (n_elements_-1)/(lblock_/(key_size+1))+1, 1), dim3(key_size+1, lblock_/(key_size+1), 1), 0, stream>>>(
        table, n_elements_, val_size_, ograd.dptr_ + i*data_size_*n_elements_, vals_.dptr_, matrix_.dptr_);
    }
    CHECK_EQ(hipGetLastError(), hipSuccess);

    float *pval = vals_.dptr_;
    float *pnew_val = new_vals_.dptr_;
    for (int j = 0; j <= key_size; ++j) {
      blur<key_size><<<dim3((n_keys_-1)/lblock_+1, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, val_size_, j, pval, pnew_val, matrix_.dptr_);
      CHECK_EQ(hipGetLastError(), hipSuccess);
      std::swap(pval, pnew_val);
    }

    if (param_.normalize) {
      slice<key_size, true><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size_, pval, data_grad.dptr_ + i*data_size_*n_elements_, matrix_.dptr_);
    } else {
      slice<key_size, false><<<dim3(nblock_, 1, 1), dim3(lblock_, 1, 1), 0, stream>>>(
        table, n_elements_, val_size_, pval, data_grad.dptr_ + i*data_size_*n_elements_, matrix_.dptr_);
    }
    CHECK_EQ(hipGetLastError(), hipSuccess);
  }
}

  
template<>
Operator *CreateOp<gpu>(PermutohedralParam param, int key_size) {
  switch (key_size) {
   case 2: return new CuPermutohedralOp<2>(param);
   case 3: return new CuPermutohedralOp<3>(param);
   case 4: return new CuPermutohedralOp<4>(param);
   case 5: return new CuPermutohedralOp<5>(param);
   case 6: return new CuPermutohedralOp<6>(param);
   case 7: return new CuPermutohedralOp<7>(param);
   case 8: return new CuPermutohedralOp<8>(param);
   case 9: return new CuPermutohedralOp<9>(param);
   case 10: return new CuPermutohedralOp<10>(param);
   case 11: return new CuPermutohedralOp<11>(param);
   case 12: return new CuPermutohedralOp<12>(param);
   case 13: return new CuPermutohedralOp<13>(param);
   case 14: return new CuPermutohedralOp<14>(param);
   case 15: return new CuPermutohedralOp<15>(param);
   case 16: return new CuPermutohedralOp<16>(param);
   default:
    LOG(FATAL) << "GPU not supported";
    return NULL;
  }
}

}  // namespace op
}  // namespace mxnet

